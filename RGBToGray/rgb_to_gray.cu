#include "hip/hip_runtime.h"
#include "rgb_to_gray.cuh"
#include <>

__global__ void rgb2grayincuda(uchar3 *const d_in, unsigned char *const d_out, uint imgheight, uint imgwidth)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < imgwidth && idy < imgheight)
    {
        uchar3 rgb = d_in[idy * imgwidth + idx];
        d_out[idy * imgwidth + idx] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
    }
}

void Rgb2Gray(const cv::Mat &src, cv::Mat &gray)
{
    const uint imgheight = src.rows;
    const uint imgwidth = src.cols;

    uchar3 *d_in;
    unsigned char *d_out;

    hipMalloc((void**)&d_in, imgwidth * imgheight * sizeof(uchar3));
    hipMalloc((void**)&d_out, imgheight * imgwidth * sizeof(unsigned char));

    hipMemcpy(d_in, src.data, imgwidth * imgheight * sizeof(uchar3), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32); // 注意：测试机器GPU block所含的最大thread数量为1024，超过后核函数将不会被调用
    dim3 blocksPerGrid((imgwidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (imgheight + threadsPerBlock.y - 1) / threadsPerBlock.y);
    rgb2grayincuda<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, imgheight, imgwidth);
    hipDeviceSynchronize();

    gray = cv::Mat::zeros(imgheight, imgwidth, CV_8UC1);
    hipMemcpy(gray.data, d_out, imgwidth * imgheight * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
}